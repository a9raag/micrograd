#include "hip/hip_runtime.h"
#include <iostream> 
#include "include/compute1d.h"
#include "cuda_compute.cu"
#include <stdexcept>
using namespace std;

template <typename T>
Compute1D<T>::Compute1D() {
}

template <typename T>
Compute1D<T>::~Compute1D() {
    hipFree(this->data);
}

template <typename T>
T* Compute1D<T>::getData() {
    return this->data;
}

template <typename T>
void Compute1D<T>::setData(T* data) {
    this->data = data;
}

template <typename T>
Compute1D<T>::Compute1D(long size){
    data = new T[size];
    this->size = size;
    this->threadsPerBlock = 256;
    this->blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

    // Initialize data on GPU
    int allocSize = size * sizeof(T);
    if(hipMallocManaged(&this->data, allocSize) != hipSuccess){
        cout<<"Compute1D: Error in allocating memory"<<endl;
        cout<<hipGetErrorString(hipGetLastError())<<endl;
        throw invalid_argument("Error in allocating memory");
    }
    hipDeviceSynchronize();
}

template <typename T>
Compute1D<T>::Compute1D(vector<T> hdata, int size) {
    if (hipMemcpy(this->data, hdata.data(), allocSize, hipMemcpyHostToDevice) != hipSuccess) {
        cout<<"Error in copying data to GPU"<<endl;
        hipFree(this->data);
    }
    *this = Compute1D(size);
}

template <typename T>
T* Compute1D<T>::add(T* b, size_t* shape, size_t size) {
    T* c;  
    if(hipMallocManaged(&c, size * sizeof(T)) != hipSuccess){
        cout<<"Error in allocating memory"<<endl;
        throw runtime_error("Error in allocating memory");
    }
    addKernel<<<blocksPerGrid, threadsPerBlock>>>(this->data, b, c, size);
    hipDeviceSynchronize();
    return c;
}

template <typename T>
T* Compute1D<T>::add(double b, size_t* shape, size_t size) {
    T* c;  
    if(hipMallocManaged(&c, size * sizeof(T)) != hipSuccess){
        cout<<"Error in allocating memory"<<endl;
        throw runtime_error("Error in allocating memory");
    }
    addKernel<<<blocksPerGrid, threadsPerBlock>>>(this->data, b, c, size);
    hipDeviceSynchronize();
    return c;
}

template <typename T>
T* Compute1D<T>::dot(T* b, size_t* shape, size_t size){ 
    dim3 blockDim(16, 16);
    dim3 gridDim((shape[0] + blockDim.x - 1) / blockDim.x, (shape[1] + blockDim.y - 1) / blockDim.y);
    T *c; 
    
    if(hipMallocManaged(&c, size * sizeof(T)) != hipSuccess){
        cout<<"1dcompute:dot: Error in allocating memory"<<endl;
        throw runtime_error("1dcompute:dot Error in allocating memory");
    }
    dotKernel<<<blocksPerGrid, threadsPerBlock>>>(this->data, b, c, size);
    // dotKernel2d<<<gridDim, blockDim>>>(this->data, b, c, shape[0], shape[1]);
    hipDeviceSynchronize();
    return c;

}

template <typename T>
T* Compute1D<T>::dot(double b, size_t* shape, size_t size){ 
    T *c; 
    
    if(hipMallocManaged(&c, size * sizeof(T)) != hipSuccess){
        cout<<"Error in allocating memory"<<endl;
        throw runtime_error("Error in allocating memory");
    }
    dotKernel<<<blocksPerGrid, threadsPerBlock>>>(this->data, b, c, size);
    hipDeviceSynchronize();
    return c;

}

template <typename T>
T* Compute1D<T>::mul(T* b, size_t* shape, size_t size){ 
    T* c; 
    if(hipMallocManaged(&c, size * sizeof(T)) != hipSuccess){
        cout<<"Error in allocating memory"<<endl;
        throw runtime_error("Error in allocating memory");
    }
    mulKernel<<<blocksPerGrid, threadsPerBlock>>>(this->data, b, c, size);
    hipDeviceSynchronize();
    return c;
}

template <typename T>
T* Compute1D<T>::mul(double b){ 
    T* c; 
    if(hipMallocManaged(&c, size * sizeof(T)) != hipSuccess){
        cout<<"Error in allocating memory"<<endl;
        throw runtime_error("Error in allocating memory");
    }
    mulKernel<<<blocksPerGrid, threadsPerBlock>>>(this->data, b, c, size);
    hipDeviceSynchronize();
    return c;
}

template <typename T>
T* Compute1D<T>::pow(double n){ 
    T* out; 
    if(hipMallocManaged(&out, size * sizeof(T)) != hipSuccess){
        cout<<"Error in allocating memory"<<endl;
        throw runtime_error("Error in allocating memory");
    }
    powKernel<<<blocksPerGrid, threadsPerBlock>>>(this->data, out, n, size);
    hipDeviceSynchronize();
    return out;
}

template <typename T>
T* Compute1D<T>::tanh(){ 
    T* out; 
    if(hipMallocManaged(&out, size * sizeof(T)) != hipSuccess){
        cout<<"Error in allocating memory"<<endl;
        throw runtime_error("Error in allocating memory");
    }
    tanhKernel<<<blocksPerGrid, threadsPerBlock>>>(this->data, out, size);

    hipDeviceSynchronize();
    return out;
}

template <typename T>
void Compute1D<T>::fill(T val){
    fillKernel<<<blocksPerGrid, threadsPerBlock>>>(this->data, val, size);
    hipDeviceSynchronize();
}
template <typename T>
void Compute1D<T>::fillRandom(unsigned int seed){
    fillRandomKernel<<<blocksPerGrid, threadsPerBlock>>>(this->data, size, seed);
    hipDeviceSynchronize();
}