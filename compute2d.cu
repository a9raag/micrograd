#include "hip/hip_runtime.h"
#include <iostream> 
#include "include/compute2d.h"
// #include "include/cuda_compute.h"
// #include "cuda_compute.cu"
#include <stdexcept>
#include "compute2d.h"
using namespace std;

template <typename T>
Compute2D<T>::Compute2D(){
};

template <typename T>
Compute2D<T>::~Compute2D(){
    hipFree(this->data);
}

template <typename T>
void Compute2D<T>:: setData(T* data){
    this->data = data;
}
template <typename T>
T* Compute2D<T>::getData(){
    return this->data;
}

template <typename T>
Compute2D<T>::Compute2D(int x, int y){
    this->data = new T[x * y];
    this->size = x * y;
    this->shape[0] = x;
    this->shape[1] = y;
    int allocSize = this->size * sizeof(T);

    this->block = dim3(x, y);
    this->grid = dim3((x + this->block.x - 1) / this->block.x, (y + this->block.y - 1) / this->block.y);
    
    if(hipMallocManaged(&this->data, allocSize) != hipSuccess){
        cout<<"Compute2D: Error in allocating memory"<<endl;
        // TODO: Add error message
        cout<<hipGetErrorString(hipGetLastError())<<endl;
        throw runtime_error("Error in allocating memory");
    }
    hipDeviceSynchronize();
    
}

template <typename T>
T* Compute2D<T>::add(BaseCompute<T>& compute){
    // if (size != this->size){
    //     throw invalid_argument("Size of the two arrays must be the same");
    // }
    T* result = new T[size];
    if(hipMallocManaged(&result, size * sizeof(T)) != hipSuccess){
        cout<<"Error in allocating memory"<<endl;
        cout<<hipGetErrorString(hipGetLastError())<<endl;
        throw runtime_error("Error in allocating memory");
    }
    addKernel2d<<<this->grid, this->block>>>(this->data, compute.getData(), result, shape[0], shape[1]);
    hipDeviceSynchronize();
    return result;
}

template <typename T>
T*  Compute2D<T>::add(double b){
    T* result = new T[size];
    if(hipMallocManaged(&result, size * sizeof(T)) != hipSuccess){
        cout<<"Error in allocating memory"<<endl;
        cout<<hipGetErrorString(hipGetLastError())<<endl;
        throw runtime_error("Error in allocating memory");
    }
    addKernel2d<<<this->grid, this->block>>>(this->data, b, result, shape[0], shape[1]);
    hipDeviceSynchronize();
    return result;
}

template <typename T>
T* Compute2D<T>::mul(BaseCompute<T>& compute){
    if (size != this->size){
        throw invalid_argument("Size of the two arrays must be the same");
    }
    T* result = new T[size];
    if(hipMallocManaged(&result, size * sizeof(T)) != hipSuccess){
        cout<<"Error in allocating memory"<<endl;
        cout<<hipGetErrorString(hipGetLastError())<<endl;
        throw runtime_error("Error in allocating memory");
    }
    mulKernel2d<T><<<this->grid, this->block>>>(this->data, compute.getData(), result, shape[0], shape[1]);
    hipDeviceSynchronize();
    return result;
}

template <typename T>
T* Compute2D<T>::mul(double b){
    T* result = new T[size];
    if(hipMallocManaged(&result, size * sizeof(T)) != hipSuccess){
        cout<<"Error in allocating memory"<<endl;
        cout<<hipGetErrorString(hipGetLastError())<<endl;
        throw runtime_error("Error in allocating memory");
    }
    mulKernel2d<T><<<this->grid, this->block>>>(this->data, b, result, shape[0], shape[1]);
    hipDeviceSynchronize();
    return result;
}

template <typename T>
T* Compute2D<T>::dot(BaseCompute<T>& compute){
    T* result = new T[size];
    if(hipMallocManaged(&result, size * sizeof(T)) != hipSuccess){
        cout<<"Error in allocating memory"<<endl;
        cout<<hipGetErrorString(hipGetLastError())<<endl;
        throw runtime_error("Error in allocating memory");
    }
    size_t heightA = shape[0];
    size_t widthA = shape[1];
    size_t widthB = compute.getShape()[1];
    // dotKernel2d<T><<<this->grid, this->block>>>(this->data, b, result, shape[0], shape[1]);
    matrixDotProduct<<<this->grid, this->block>>>(this->data, compute.getData(), result, widthA, heightA, widthB);
    hipDeviceSynchronize();
    return result;
}

template <typename T>
T* Compute2D<T>::pow(double n){
    T* result = new T[size];
    if(hipMallocManaged(&result, size * sizeof(T)) != hipSuccess){
        cout<<"Error in allocating memory"<<endl;
        cout<<hipGetErrorString(hipGetLastError())<<endl;
        throw runtime_error("Error in allocating memory");
    }
    powKernel2d<T><<<this->grid, this->block>>>(this->data, result, n, shape[0], shape[1]);
    hipDeviceSynchronize();
    return result;
}

template <typename T>
T* Compute2D<T>::tanh(){
    T* result = new T[size];
    if(hipMallocManaged(&result, size * sizeof(T)) != hipSuccess){
        cout<<"Error in allocating memory"<<endl;
        cout<<hipGetErrorString(hipGetLastError())<<endl;
        throw runtime_error("Error in allocating memory");
    }
    tanhKernel2d<T><<<this->grid, this->block>>>(this->data, result, shape[0], shape[1]);
    hipDeviceSynchronize();
    return result;
}

template <typename T>
void Compute2D<T>::fill(T value){
    fillKernel2d<T><<<this->grid, this->block>>>(this->data, value, shape[0], shape[1]);
    hipDeviceSynchronize();
}

template <typename T>
void Compute2D<T>::fillRandom(unsigned int seed)
{
    fillRandomKernel2d<T><<<this->grid, this->block>>>(this->data, shape[0], shape[1], seed);
    hipDeviceSynchronize();
}