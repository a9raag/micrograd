#include "hip/hip_runtime.h"
#include "include/engine.h"
#include <iostream>
#include <list>
#include <math.h>
#include <random>
#include <vector>
#include <string>

using namespace std;
string to_string(double* data, int size){
    string arr = "["; 
    for(int i = 0; i < size; i++){
        arr += to_string(data[i]);
        if(i != size - 1){
            arr += ", ";
        }
    }
    arr += "]";
    return arr;
}
// default constructor
Value::Value() : data(Tensor<double>()), node_backward([](){}), prev({}), op(""), label("") {
    this->node_backward = []() {};
    this->grad = Tensor<double>();
}


Value::Value(Tensor<double> data, std::initializer_list<shared_ptr<Value>> children = {}, string op = "", string label = "")
    : data(data), node_backward([](){}), prev(children), op(op), label(label) {
        this->node_backward = []() {}; 
        this->grad = Tensor<double>(data.shape);    
};

Value::~Value() {
    // delete grad;
}
//copy constructor
Value::Value(const Value &other)
    : data(other.data), grad(other.grad), node_backward(other.node_backward), prev(other.prev), op(other.op), label(other.label) {}

//move constructor
Value::Value(Value &&other) noexcept
    : data(move(other.data)), grad(move(other.grad)), node_backward(move(other.node_backward)), prev(move(other.prev)), op(move(other.op)), label(move(other.label)) {}   


void Value::set_grad_1(){
    this->grad.fill(1.0);
}


Tensor<double> Value::getData(){
    return this->data;
}

Tensor<double> Value::getGrad(){
    return this->grad;
}

shared_ptr<Value> Value::neg(){
    Tensor<double> neg_data = Tensor<double>(this->data.shape);
    neg_data.fill(-1.0);
    auto out = make_shared<Value>(neg_data, std::initializer_list<std::shared_ptr<Value>>{}, "neg", label);
    return shared_from_this() * out;
}

shared_ptr<Value> Value::pow(const double n){
    Tensor<double> pow_data = Tensor<double>(this->data.shape);
    pow_data = this->data.pow(n);
    auto out = make_shared<Value>(pow_data, std::initializer_list<std::shared_ptr<Value>>{shared_from_this()}, "pow", label);
    out->node_backward = [this, out, n]() mutable
    {
        this->grad = this->grad +  out->grad * n * this->data.pow(n - 1);
    };
    return out;
}

shared_ptr<Value> Value::tanh(){
    Tensor<double> tanh_data = this->data.tanh();
    auto out = make_shared<Value>(tanh_data, std::initializer_list<std::shared_ptr<Value>>{shared_from_this()}, "tanh", label);
    out->node_backward = [this, out, tanh_data]() mutable
    {
        Tensor<double> ones = Tensor<double>(this->data.shape);
        ones.fill(1.0);
        auto tanh_2 = tanh_data.pow(2);
        auto tanh_grad = ones - tanh_2;
        auto grad = tanh_grad * out->grad;
        this->grad = this->grad +  grad;
    };
    return out;
}

shared_ptr<Value> Value::operator+(const double &other)
{
    Tensor<double> other_tensor = Tensor<double>(this->data.shape).fill(other);
    auto other_val = make_shared<Value>(other_tensor, std::initializer_list<std::shared_ptr<Value>>{}, "const", label);
    return shared_from_this() + other_val;
}

shared_ptr<Value> Value::operator+(const shared_ptr<Value> &other)
{
    
    auto out = make_shared<Value>(data + other->data, std::initializer_list<std::shared_ptr<Value>>{shared_from_this(), other}, "+", label);
    out->node_backward = [this, out, other]() mutable
    {  
        this->grad = this->grad +  out->grad * 1.0;
        other->grad = other->grad + out->grad * 1.0;
    };
    return out;
    
}
std::shared_ptr<Value> operator+(const std::shared_ptr<Value> &lhs, const double &rhs)
{
    return (*lhs) + rhs;
}
std::shared_ptr<Value> operator+(const std::shared_ptr<Value> &lhs, const std::shared_ptr<Value> &rhs)
{
    return (*lhs) + rhs;
}

shared_ptr<Value> Value::operator-(const double &other)
{
    Tensor<double> other_data = Tensor<double>(this->data.shape);
    other_data.fill(other);
    auto other_val = make_shared<Value>(other_data, std::initializer_list<std::shared_ptr<Value>>{}, "const", label);
    return shared_from_this() - other_val;
}

shared_ptr<Value> Value::operator-(const shared_ptr<Value> &other)
{
    return shared_from_this() + other->neg();
}

std::shared_ptr<Value> operator-(const std::shared_ptr<Value>& lhs, const std::shared_ptr<Value>& rhs) {
    return (*lhs) - rhs;
}

std::shared_ptr<Value> operator-(const std::shared_ptr<Value>& lhs, const double& rhs) {
    return (*lhs) - rhs;
}

shared_ptr<Value> Value::operator*(const double &other)
{
    Tensor<double> other_data = Tensor<double>(this->data.shape);
    other_data.fill(other);
    auto other_val = make_shared<Value>(other_data, std::initializer_list<std::shared_ptr<Value>>{}, "const", label);
    return shared_from_this() * other_val;
}

shared_ptr<Value> Value::operator*(const shared_ptr<Value> &other)
{
    auto out = make_shared<Value>(data * other->data, std::initializer_list<std::shared_ptr<Value>>{shared_from_this(), other}, "*", label);
    out->node_backward = [this, out, other]() mutable
    {
        this->grad = this->grad + out->grad * other->data;
        other->grad = other->grad + out->grad * this->data;
    };
    return out;
}
std::shared_ptr<Value> operator*(const std::shared_ptr<Value>& lhs, const std::shared_ptr<Value>& rhs) {
    return (*lhs) * rhs;
}


std::shared_ptr<Value> operator*(const std::shared_ptr<Value>& lhs, const double& rhs) {
    return (*lhs) * rhs;
}




shared_ptr<Value> Value::dot(const shared_ptr<Value> &other)
{
    auto out = make_shared<Value>(data.dot(other->data), std::initializer_list<std::shared_ptr<Value>>{shared_from_this(), other}, "dot", label);
    out->node_backward = [this, out, other]() mutable
    {
        auto otherDataT = other->data.transpose();
        this->grad = this->grad +  out->grad.dot(otherDataT);
        other->grad = other->grad + this->data.transpose().dot(out->grad);
    };
    return out;
}

shared_ptr<Value> Value::operator/(const double &other)
{
    Tensor<double> other_data = Tensor<double>(this->data.shape);
    other_data.fill(other);
    auto other_val = make_shared<Value>(other_data, std::initializer_list<std::shared_ptr<Value>>{}, "const", label);
    return shared_from_this() / other_val;
}

shared_ptr<Value> Value::operator/(const shared_ptr<Value> &other)
{
    return shared_from_this() * other->pow(-1);
}

std::shared_ptr<Value> operator/(const std::shared_ptr<Value>& lhs, const std::shared_ptr<Value>& rhs) {
    return (*lhs) / rhs;
}

std::shared_ptr<Value> operator/(const std::shared_ptr<Value>& lhs, const double& rhs) {
    return (*lhs) / rhs;
}

// String representation
std::ostream &operator<<(std::ostream &os, const shared_ptr<Value> &v)
{
    //handle null ptr 
    if(v == NULL){
        throw "v is null";
    }
    os << "Value(data=" << v->getData() << ", grad=" << v->getGrad() << ", op=" << v->op << ")";
    return os;
}

void Value::backward()
{
    std::list<shared_ptr<Value>> topo;
    std::set<shared_ptr<Value>> visited;
    std::function<void(const shared_ptr<Value>&)> build_topo = [&](const shared_ptr<Value> &v)
    {
        if (visited.count(v) == 0)
        {
            visited.insert(v);
            for (auto child : v->prev)
            {
                build_topo(child);
            }
            topo.push_front(v);
        }
    };
    this->grad.fill(1.0);

    build_topo(shared_from_this());
    for (auto v : topo)
    {
        v->node_backward();
    }
}
