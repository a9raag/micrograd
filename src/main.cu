#include "hip/hip_runtime.h"
#include <iostream>
#include<vector>
#include "tensor.cu"
#include "compute1d.cu"
#include "compute2d.cu"
#include "engine.cu"
#include <hip/hip_runtime.h>
#include "nn.cpp"
#include "data.cpp"
#include "helper.cpp"
using namespace std;

void test_compute(){
    
    vector<int> _a = {1, 2, 3};
    vector<int> _b = {2, 3, 4};

    Compute1D<int> a = Compute1D<int>(_a, _a.size());
    Compute1D<int> b = Compute1D<int>(_b, _b.size());

    int * c;
    size_t* shape = new size_t[1];
    shape[0] = 3;
    c = a.add(b);

    cout<<"C:"<<endl;
    for(int i = 0; i < _a.size(); i++){
        cout<<c[i]<<endl;
    }
}

void test_sub_tensor(){
    cout<<"=========================="<<endl;
    cout<<"START: Test SubTensor"<<endl;
    cout<<"=========================="<<endl;
    Tensor<float> a = Tensor<float>({3, 3});
    a(0,0) = 1;
    a(0,1) = 2;
    a(0,2) = 3;
    a(1,0) = 4;
    a(1,1) = 5;
    a(1,2) = 6;
    a(2,0) = 7;
    a(2,1) = 8;
    a(2,2) = 9;
    cout<<"a"<<endl;
    cout<<a<<endl;
    auto b = a.subTensor({{0, 1}, {0, 3}});
    cout<<"b"<<endl;
    cout<<b<<endl;
    cout<<"END: Test SubTensor"<<endl;

}

void test_tensor_1d(){
    cout<<"=========================="<<endl;
    cout<<"START: Testing Tensor 1d"<<endl;
    cout<<"=========================="<<endl;

    Tensor<float> a({3}), b({3});
    a(0) = 1;
    a(1) = 2; 
    a(2) = 3;

    b(0) = 4;
    b(1) = 5;
    b(2) = 6;

    cout<<"a:"<<endl;
    cout<<a<<endl;

    cout<<"b:"<<endl;
    cout<<b<<endl;

    auto c = a+b;
    cout<<"c:"<<endl;
    cout<<c<<endl;

    c = a * b;
    cout<<"c:"<<endl;
    cout<<c<<endl;

    auto d = c - a;
    cout<<"c:"<<endl;
    cout<<d<<endl;

    d = b / a;
    cout<<"b/a"<<endl;
    cout<<d<<endl;

    cout<<"a**2"<<endl;
    cout<<a.pow(2.0)<<endl;

    d = a.dot(b);
    cout<<"a@b"<<endl;
    cout<<d<<endl;

    Tensor<float> e = Tensor<float>({1});
    e(0) = 2.0;
    d = d + e;
    cout<<"d+e"<<endl;
    cout<<d<<endl;

    cout<<"END: Testing Tensor 1d"<<endl;
}
void test_tensor(){
    cout<<"=========================="<<endl;
    cout<<"START: Test Tensor: a is 3x3, b is 3x3"<<endl;
    cout<<"=========================="<<endl;
    Tensor<float> a({3, 3});
    Tensor<float> b({3, 3});
    a(0,0) = 1;
    a(0,1) = 2;
    a(0,2) = 3;

    a(1,0) = 4;
    a(1,1) = 5;
    a(1,2) = 6;

    a(2,0) = 7;
    a(2,1) = 8;
    a(2,2) = 9;

    b(0,0) = 20;
    b(0,1) = 30;
    b(0,2) = 40;

    b(1,0) = 50;
    b(1,1) = 60;
    b(1,2) = 70;

    b(2,0) = 80;
    b(2,1) = 90;
    b(2,2) = 100;
    cout<<"a:"<<endl;
    cout<<a<<endl;
    cout<<"b:"<<endl;
    cout<<b<<endl;

    auto c = a+b; 
    cout<<"c:"<<endl;
    cout << c << endl;

    c = a * b;
    cout<<"c:"<<endl;
    cout << c <<endl;

    auto d = c - a;
    cout<<"c:"<<endl;
    cout << d << endl;
    
    d =  b / a; 
    cout<<"b/a"<<endl;
    cout << d << endl;

    cout<<"a**2"<<endl;
    cout<<a.pow(2.0)<<endl;

    cout<<"a.tanh()"<<endl;
    cout<<a.tanh()<<endl;
    cout<<"END: Testing Tensor"<<endl;
    
}
void test_tensor_2d(){
    cout<<"=========================="<<endl;
    cout<<"START: Test Tensor 2d: a is 2x3, b is 3x2"<<endl;
    cout<<"=========================="<<endl;
    Tensor<float> a({2, 3});
    Tensor<float> b({3, 2});
    a(0, 0) = 1;
    a(0, 1) = 2;
    a(0, 2) = 3;
    a(1, 0) = 4;
    a(1, 1) = 5;
    a(1, 2) = 6;


    b(0, 0) = 20;
    b(0, 1) = 30;
    b(1, 0) = 40;
    b(1, 1) = 50;
    b(2, 0) = 60;
    b(2, 1) = 70;
    
    cout<<"a:"<<endl;
    cout<<a<<endl;
    cout<<"b:"<<endl;
    cout<<b<<endl;

    // auto c = a+b;
    // cout<<"c:a+b"<<endl;
    // cout<<c<<endl;

    // c = a * b;
    // cout<<"c:a*b"<<endl;
    // cout<<c<<endl;

    auto c = a.dot(b);
    cout<<"c:a@b"<<endl;    
    cout<<c<<endl;

    cout<<"END: Test Tensor 2d"<<endl;

}
void test_value2d(){
    cout<<"=========================="<<endl;
    cout<<"START: Test Value 2d: a is 3x3, b is 3x3"<<endl;
    cout<<"=========================="<<endl;
    Tensor<float> a({3, 3}), b({3, 3});

    a(0,0) = 1;
    a(0,1) = 2;
    a(0,2) = 3;

    a(1,0) = 4;
    a(1,1) = 5;
    a(1,2) = 6;

    a(2,0) = 7;
    a(2,1) = 8;
    a(2,2) = 9;

    b(0,0) = 10;
    b(0,1) = 20;
    b(0,2) = 30;

    b(1,0) = 40;
    b(1,1) = 50;
    b(1,2) = 60;

    b(2,0) = 70;
    b(2,1) = 80;
    b(2,2) = 90;
    cout<<"a"<<endl;
    cout<<a<<endl;
    cout<<"--------------------------"<<endl;

    cout<<"b"<<endl;
    cout<<b<<endl;
    cout<<"--------------------------"<<endl;

    auto c = a + b;
    cout<<"c: a + b"<<endl;
    cout<<c<<endl;
    cout<<"--------------------------"<<endl;

    auto d = a * b;
    cout<<"d: a * b"<<endl;
    cout<<d<<endl;
    cout<<"--------------------------"<<endl;

    shared_ptr<Value> val_a = std::make_shared<Value>(a);
    shared_ptr<Value> val_b = std::make_shared<Value>(b);

    auto val_c = val_a + val_b;
    cout<<"val_c: val_a + val_b"<<endl;
    cout<<val_c->getData()<<endl;
    cout<<"--------------------------"<<endl;

    auto val_d = val_a * val_b;
    cout<<"val_d: val_a * val_b"<<endl;
    cout<<val_d->getData()<<endl;
    cout<<"--------------------------"<<endl;

    auto val_e = val_a->dot(val_b);
    cout<<"val_e: val_a.dot(val_b)"<<endl;
    cout<<val_e->getData()<<endl;
    cout<<"--------------------------"<<endl;


    val_c = val_c->pow(0.5);
    cout<<"val_c: val_c ** 0.5"<<endl;
    cout<<val_c->getData()<<endl;
    cout<<"--------------------------"<<endl;

    val_c = val_c->tanh();
    cout<<"val_c: val_c.tanh()"<<endl;
    cout<<val_c->getData()<<endl;
    cout<<"--------------------------"<<endl;

    val_c->set_grad_1();
    val_c->node_backward();
    cout<<"val_a->grad:"<<endl;
    cout<<val_a->getGrad()<<endl;
    cout<<"--------------------------"<<endl;

    // test mean 
    Tensor<float> e = Tensor<float>({5, 1}) ;
    e(0, 0) = 1;
    e(1, 0) = 2;
    e(2, 0) = 3;
    e(3, 0) = 4;
    e(4, 0) = 5;

    Tensor<float> f = Tensor<float>({5, 1});
    f(0, 0) = 11;
    f(1, 0) = 22;
    f(2, 0) = 33;
    f(3, 0) = 44;
    f(4, 0) = 55;

    
    val_e = std::make_shared<Value>(e);
    auto val_f = val_e->mean();
    cout<<"val_f: val_e.mean()"<<endl;
    cout<<val_f->getData()<<endl;

    val_f  = make_shared<Value>(f);

    auto val_g = val_f - val_e;
    
    cout<<"val_g: val_f - val_e"<<endl;
    cout<<val_g->getData()<<endl;

    val_g = val_g->mean();  
    cout<<"val_g: val_g.mean()"<<endl;
    cout<<val_g->getData()<<endl;


    cout<<"END: Test Value 2d: a is 3x3, b is 3x3"<<endl;
    

}

void test_matrix_vector_ops(){
    cout<<"=========================="<<endl;
    cout<<"START: Test Matrix Vector Ops"<<endl;
    cout<<"=========================="<<endl;
    Tensor<float> a = Tensor<float>({3, 3});
    Tensor<float> b = Tensor<float>({3, 1});

    a(0,0) = 1;
    a(0,1) = 2;
    a(0,2) = 3;
    a(1,0) = 4;
    a(1,1) = 5;
    a(1,2) = 6;
    a(2,0) = 7;
    a(2,1) = 8;
    a(2,2) = 9;

    b(0,0) = 10;
    b(1,0) = 20;
    b(2,0) = 30;
    
    cout<<"a"<<endl;
    cout<<a<<endl;
    cout<<"b"<<endl;
    cout<<b<<endl;

    auto c = a + b; 
    cout<<"c = a+b"<<endl;
    cout<<c<<endl;

    c = a * b;
    cout<<"c = a * b"<<endl;
    cout<<c<<endl;

    auto d = c - a;
    cout<<"d = c - a"<<endl;
    cout<<d<<endl;

    // d = b / a;
    // cout<<"d = b/a"<<endl;
    // cout<<d<<endl;

    Tensor<float> vec_a = Tensor<float>({3});
    vec_a(0) = 1;
    vec_a(1) = 2;
    vec_a(2) = 3;
    cout<<"vec_a"<<endl;
    cout<<vec_a<<endl;
    c = a + vec_a; 
    cout<<"c = a + vec_a"<<endl;
    cout<<c<<endl;
    cout<<"--------------------------"<<endl;
    
}   

void test_backprop(){
    cout<<"=========================="<<endl;
    cout<<"START: Test Backprop"<<endl;
    cout<<"=========================="<<endl;
    cout << "Starting forward pass" << endl;
    Tensor<float> xt1 = Tensor<float>({1});
    xt1(0) = 2.0;
    shared_ptr<Value> x1 = std::make_shared<Value>(xt1);
    Tensor<float> xt2 = Tensor<float>({1});
    xt2(0) = 0.0;
    shared_ptr<Value> x2 = std::make_shared<Value>(xt2);
    x1->label = "x1";
    x2->label = "x2";
    Tensor<float> wt1 = Tensor<float>({1});
    wt1(0) = -3.0;
    shared_ptr<Value> w1 = std::make_shared<Value>(wt1);
    Tensor<float> wt2 = Tensor<float>({1});
    wt2(0) = 0.0;
    shared_ptr<Value> w2 = std::make_shared<Value>(wt2);
    w1->label = "w1";
    w2->label = "w2";

    Tensor<float> bt = Tensor<float>({1});
    bt(0) = 6.8813735870195432;
    shared_ptr<Value> b = std::make_shared<Value>(bt);
    b->label = "b";
    shared_ptr<Value> x1w1 = x1 * w1;
    x1w1->label = "x1w1";
    shared_ptr<Value> x2w2 = x2 * w2;
    x2w2->label = "x2w2";
    shared_ptr<Value> x1w1x2w2 = x1w1 + x2w2;
    x1w1x2w2->label = "x1w1x2w2";
    shared_ptr<Value> n = x1w1x2w2 + b;
    n->label = "n";
    shared_ptr<Value> o = n->tanh();
    o->label = "o";

    // Value c = o.pow(2.0);
    // c.label = 'c';

    std::cout << "Starting backward pass" << std::endl;
    o->backward();
    // cout << "c: " << c << endl;
    // cout<<"c: "<<c.grad<<endl;
    cout << "o: " << o << endl;
    cout << "n: " << n << endl;
    cout << "x1w1x2w2: " << x1w1x2w2 << endl;
    cout << "x2w2: " << x2w2<< endl;
    cout << "x1w1: " << x1w1 << endl;
    cout << "b: " << b << endl;
    cout << "w2: " << w2 << endl;
    cout << "x2: " << x2 << endl;
    cout << "w1: " << w1 << endl;
    cout << "x1: " << x1 << endl;
    cout<<"END: Test Backprop"<<endl;
}
void test_random(){
    cout<<"=========================="<<endl;
    cout<<"START: Test Random"<<endl;
    cout<<"=========================="<<endl;
    Tensor<float> a = Tensor<float>({3, 3}).randomize();
    
    cout<<"a"<<endl;
    cout<<a.randomize()<<endl;

    Tensor<float> b ({10});
    b.randomize();
    cout<<"b"<<endl;
    cout<<b<<endl;

    Tensor<float> c ({1, 10});
    c.randomize();
    cout<<"c"<<endl;
    cout<<c<<endl;
    cout<<"END: Test Random"<<endl;
}

void test_value_broadcast(){
    cout<<"=========================="<<endl;
    cout<<"START: Test Value Broadcast"<<endl;
    cout<<"=========================="<<endl;
    Tensor<float> a = Tensor<float>({3, 3});
    Tensor<float> b = Tensor<float>({3, 3});
    a(0,0) = 1;
    a(0,1) = 2;
    a(0,2) = 3;
    a(1,0) = 4;
    a(1,1) = 5;
    a(1,2) = 6;
    a(2,0) = 7;
    a(2,1) = 8;
    a(2,2) = 9;

    b(0,0) = 10;
    b(0,1) = 20;
    b(0,2) = 30;
    b(1,0) = 40;
    b(1,1) = 50;
    b(1,2) = 60;
    b(2,0) = 70;
    b(2,1) = 80;
    b(2,2) = 90;
    



    cout<<"a"<<endl;
    cout<<a<<endl;
    cout<<"b"<<endl;
    cout<<b<<endl;
    auto c = a + b;
    cout<<"c"<<endl;
    cout<<c<<endl;

    auto val_a = std::make_shared<Value>(a);
    auto val_b = std::make_shared<Value>(b);
    auto val_c = val_a + 1.0;
    cout<<"val_c: val_a + 1.0"<<endl;
    cout<<val_c->getData()<<endl;

    Tensor<float> row = Tensor<float>({1, 3});
    row(0,0) = 1;
    row(0,1) = 2;
    row(0,2) = 3;
    auto val_row = std::make_shared<Value>(row);
    cout<<"val_row: "<<val_row->getData()<<endl;
    auto val_d = val_a + val_row * 10;
    cout<<"val_d: val_a + val_row"<<endl;
    cout<<val_d->getData()<<endl;

    Tensor<float> column = Tensor<float>({3, 1});
    column(0,0) = 100;
    column(1,0) = 200;
    column(2,0) = 300;

    auto val_column = std::make_shared<Value>(column);
    cout<<"val_column: \n"<<val_column->getData()<<endl;
    auto val_e = val_a + val_column;
    cout<<"val_e: val_a + val_column"<<endl;
    cout<<val_e->getData()<<endl;

    auto val_f = val_a->dot(val_column);
    cout<<"val_f: val_a.dot(val_column)"<<endl;
    cout<<val_f->getData()<<endl;

    cout<<"END: Test Value Broadcast"<<endl;

}
void test_gradient(){
    cout<<"=========================="<<endl;
    cout<<"START: Test Gradient"<<endl;
    cout<<"=========================="<<endl;
    Tensor<float> a = Tensor<float>({3, 3});
    Tensor<float> b = Tensor<float>({3, 3});
    a(0,0) = 1;
    a(0,1) = 2;
    a(0,2) = 3;
    a(1,0) = 4;
    a(1,1) = 5;
    a(1,2) = 6;
    a(2,0) = 7;
    a(2,1) = 8;
    a(2,2) = 9;

    b(0,0) = 10;
    b(0,1) = 20;
    b(0,2) = 30;
    b(1,0) = 40;
    b(1,1) = 50;
    b(1,2) = 60;
    b(2,0) = 70;
    b(2,1) = 80;
    b(2,2) = 90;
    cout<<"a"<<endl;
    cout<<a<<endl;
    cout<<"b"<<endl;
    cout<<b<<endl;

    auto val_a = std::make_shared<Value>(a);
    auto val_b = std::make_shared<Value>(b);
    auto val_c = val_a + val_b;
    cout<<"val_c: val_a + val_b"<<endl;
    cout<<val_c->getData()<<endl;

    val_c->set_grad_1();
    val_c->backward();
    cout<<"val_a grad"<<endl;
    cout<<val_a->getGrad()<<endl;
    cout<<"val_b grad"<<endl;
    cout<<val_b->getGrad()<<endl;

    
    val_a = std::make_shared<Value>(a);
    val_b = std::make_shared<Value>(b);
    auto val_d = val_a->dot(val_b);
    cout<<"val_d: val_a @ val_b"<<endl;
    cout<<val_d->getData()<<endl;

    val_d->set_grad_1();
    val_d->backward();
    cout<<"val_a grad"<<endl;
    cout<<val_a->getGrad()<<endl;
    cout<<"val_b grad"<<endl;
    cout<<val_b->getGrad()<<endl;


    val_a = std::make_shared<Value>(a);
    val_b = std::make_shared<Value>(b);
    val_d = val_a * val_b;
    cout<<"val_d: val_a * val_b"<<endl;
    cout<<val_d->getData()<<endl;

    val_d->set_grad_1();
    val_d->backward();
    cout<<"val_a grad"<<endl;
    cout<<val_a->getGrad()<<endl;
    cout<<"val_b grad"<<endl;
    cout<<val_b->getGrad()<<endl;

    cout<<"END: Test Gradient"<<endl;

}
void test_layer(){
    cout<<"=========================="<<endl;
    cout<<"START: Test Layer"<<endl;
    cout<<"=========================="<<endl;
    Tensor<float> a = Tensor<float>({1, 4});
    a(0, 0) = 2.0;
    a(0, 1) = 3.0;
    a(0, 2) = 4.0;
    a(0, 3) = 1.0;

    Layer layer = Layer({4, 2});
    shared_ptr<Value> x = std::make_shared<Value>(a);
    auto out = layer(x);
    out->set_grad_1();
    out->backward();
    cout<<"out: "<<out<<endl;
    cout<<"out grad: "<<out->getGrad()<<endl;
    cout<<"END: Test Layer"<<endl;
}

void test_mlp(){
    cout<<"=========================="<<endl;
    cout<<"START: Test MLP"<<endl;
    cout<<"=========================="<<endl;
    Tensor<float> a = Tensor<float>({1, 4});
    a(0, 0) = 2.0;
    a(0, 1) = 3.0;
    a(0, 2) = 4.0;
    a(0, 3) = 1.0;

    MLP mlp = MLP(4, {4, 2, 1});
    shared_ptr<Value> x = std::make_shared<Value>(a);
    auto out = mlp(x);
    out->set_grad_1();
    out->backward();
    cout<<"out: "<<out<<endl;
    cout<<"out grad: "<<out->getGrad()<<endl;
    cout<<"END: Test MLP"<<endl;

}

void test_large_mlp(){
    cout<<"=========================="<<endl;
    cout<<"START: Test Large MLP"<<endl;
    cout<<"=========================="<<endl;
    vector<vector<float>> xs = {
        {2.0, 3.0, -1.0},
        {3.0, -1.0, 0.5},
        {0.5, 1.0, 1.0},
        {1.0, 1.0, -1.0}
    };
    vector<vector<float>> ys = {
        {1.0},
        {-1.0},
        {-1.0},
        {1.0}
    };
    Tensor<float> x = Tensor<float>({4, 3});
    Tensor<float> y = Tensor<float>({4, 1});
    for(int i = 0; i < 4; i++){
        for(int j = 0; j < 3; j++){
            x(i, j) = xs[i][j];
        }
        y(i, 0) = ys[i][0];
    }
    MLP mlp = MLP(3, {4, 4, 1});
    shared_ptr<Value> x_val = std::make_shared<Value>(x);
    shared_ptr<Value> y_val = std::make_shared<Value>(y);
    

    int epoch = 20; 
    float lr = 0.01;


    for(int i = 0; i < epoch; i++){
        auto out = mlp(x_val);
        // out = out->tanh();
        auto loss = out - y_val;
        loss = loss->pow(2.0);
        loss = loss->mean();
        mlp.zero_grad();
        loss->set_grad_1();
        loss->backward();
        mlp.update_params(lr);
        if(i%10 == 0){
            cout<<"epoch: "<<i<<"/"<<epoch<<", loss: "<<loss->getData()<<endl;
            cout<<"out: "<<out->getData()<<endl;
            cout<<"y: "<<y_val->getData()<<endl;
        }

    }
    cout<<"out: "<<mlp(x_val)<<endl;
    cout<<"y: "<<y_val<<endl;
    cout<<"END: Test Large MLP"<<endl;
}

void test_data(){
    cout<<"=========================="<<endl;
    cout<<"START: Test Data"<<endl;
    cout<<"=========================="<<endl;

    Data data("../names.txt");
    vector<string> words = data.getWords();
    cout <<"Words size: " << words.size() << endl;
    cout << "Vocab Size: "<< data.getVocabSize() << endl;
    map<char, int> stoi = data.getStoi();
    map<int, char> itos = data.getItos();

    cout<<"stoi = ";

    cout<< "{";
    for(auto kv: stoi){
        cout << kv.first << ":" << kv.second << ", ";
    }
    cout << "}";
    cout << endl;

    cout<<"itos = ";
    cout << "{";
    for(auto kv: itos){
        cout << kv.first << ":" << kv.second << ", ";

    }
    cout << "}";
    cout << endl;

    cout<<"END: Test Data"<<endl;


}

void train_bigram_probability(){
    // intialise data
    Data data("../names.txt");
    vector<string> words = data.getWords();
    cout <<"Words size: " << words.size() << endl;
    cout << "Vocab Size: "<< data.getVocabSize() << endl;
    map<char, int> stoi = data.getStoi();
    map<int, char> itos = data.getItos();

    // intialise tensors 
    Tensor<float> N = Tensor<float>({data.getVocabSize(), data.getVocabSize()}).fill(0.0);
    hipDeviceSynchronize();
    cout<<"START: Fill N with bigram counts"<<endl;
    for(auto word : words){
        word = "." + word + ".";
        for(int i = 0; i < word.size()-1; i++){
            int c1 = stoi[word[i]];
            int c2 = stoi[word[i+1]];
            N(c1, c2) += 1;
        }
    }
    cout<<"END: Fill N with bigram counts"<<endl;

    cout<<"N: "<<endl;
    cout<<N<<endl;

    auto P = N + 1.0;
    cout<<"P.sum(1) "<<P.sum(1)<<endl;
    P = P * P.sum(1).pow(-1);
    
    auto loss = P.log();
    shared_ptr<Value> loss_val = std::make_shared<Value>(loss)->mean();
    cout<<"loss: "<<loss_val->getData()<<endl;
}

void train_bigram_nn(){
    Data data("/home/anurag/dev/micrograd/names.txt");
    vector<string> words = data.getWords();
    cout <<"Words size: " << words.size() << endl;
    cout << "Vocab Size: "<< data.getVocabSize() << endl;
    map<char, int> stoi = data.getStoi();
    map<int, char> itos = data.getItos();


    Tensor<float> N = Tensor<float>({data.getVocabSize(), data.getVocabSize()});
    N.fill(0.0);
    vector<float> xsv; 
    vector<float> ysv;
    cout<<"START: Create input and output data"<<endl;
    
    for(auto word : words){
        word = "." + word + ".";
        for(int i = 0; i < word.size()-1; i++){
            int c1 = stoi[word[i]]; 
            int c2 = stoi[word[i+1]];
            xsv.push_back(c1);
            ysv.push_back(c2);
        }
    }
    Tensor<float> xs = Tensor<float>({xsv.size()});
    xs.setData(xsv.data());
    Tensor<float> ys = Tensor<float>({ysv.size()});
    ys.setData(ysv.data());


    auto xenc = oneHot(xs, data.getVocabSize());
    auto w = Tensor<float>({data.getVocabSize(), data.getVocabSize()}).randomize();

    shared_ptr<Value> x = std::make_shared<Value>(xenc);
    shared_ptr<Value> y = std::make_shared<Value>(ys);
    shared_ptr<Value> w_val = std::make_shared<Value>(w);


    int epoc = 100;
    float lr = 0.01;
    vector<vector<size_t>> indices = {{}}; 
    for(int i = 0; i < xsv.size(); i++){
        indices[0].push_back(i);
        indices[1].push_back((size_t)ysv[i]);

    }

    for(int i = 0; i < epoc; i++){
        auto logits = x->dot(w_val);
        cout<<"logits shape: " << logits->getData().shape[0] << " " << logits->getData().shape[1] << endl;
        auto counts = logits->exp();
        auto temp_count = counts->sum(1)->pow(-1);
        cout<<"temp count shape :"<<temp_count->getData().shape[0] << " " << temp_count->getData().shape[1] << endl;
        cout<<temp_count->subTensor({{1, 2}, {0, 10}})<<endl;
        auto probs = counts / temp_count; 

        Tensor<float> targetProbs = probs->getData().fancyIndexing(indices);
        auto loss = make_shared<Value>(targetProbs)->log()->mean();

        w_val->zero_grad();

        loss->set_grad_1();
        loss->backward();

        w_val->setData(w_val->getData() - w_val->getGrad() * lr);
        if(i%10 == 0){
            cout<<"epoch: "<<i<<"/"<<epoc<<", loss: "<<loss->getData()<<endl;
        }
    }
    

}
int main(int argc, char const *argv[]){
    // test_tensor_1d();
    test_tensor_2d();
    // test_value2d();
    // test_backprop();
    // test_gradient();
    // test_random();
    // test_matrix_vector_ops();
    // test_value_broadcast();
    // test_layer();
    // test_mlp();
    // test_large_mlp();
    // test_sub_tensor();  
    train_bigram_nn();   
    return 0;

}