#include "hip/hip_runtime.h"
#include <iostream> 
#include "include/compute1d.h"
#include "cuda_compute.cu"
#include <thrust/device_vector.h>

#include <stdexcept>
#include "compute1d.h"


using namespace std;

template <typename T>
Compute1D<T>::Compute1D() {
    this->size = 0;
    this->data = nullptr;
}

template <typename T>
Compute1D<T>::~Compute1D() {
    hipFree(this->data);
}

template <typename T>
T* Compute1D<T>::getData() {

    return this->data;
}

template <typename T>
void Compute1D<T>::setData(T* data) {
    hipMemcpy(this->data, data, this->size * sizeof(T), hipMemcpyHostToDevice);
}

template <typename T> 
void Compute1D<T>::allocateMemory(T* data, size_t size) {
    hipDeviceProp_t prop;
    int deviceId;
    hipGetDevice(&deviceId); // Get current device ID
    hipGetDeviceProperties(&prop, deviceId); // Get device properties
    this->threadsPerBlock = prop.maxThreadsPerBlock;
    // int blockSize = (maxThreadsPerBlock < 256) ? maxThreadsPerBlock : 256; // Example choice, considering hardware limit

    this->blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    this->size = size;
    if(hipMallocManaged(&this->data, size * sizeof(T)) != hipSuccess){
        cout<<"Error in allocating memory"<<endl;
        throw runtime_error("Error in allocating memory");
    }
    if (hipMemcpy(this->data, data, size * sizeof(T), hipMemcpyHostToDevice) != hipSuccess) {
        cout<<"Error in copying data to GPU"<<endl;
        hipFree(this->data);
    }
}

template <typename T>
Compute1D<T>::Compute1D(size_t size)
{
    this->data = new T[size];
    this->shape[0] = size;
    this->size = size;
    this->threadsPerBlock = 32;
    this->blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

    // Initialize data on GPU
    int allocSize = this->size * sizeof(T);
    if(hipMallocManaged(&this->data, allocSize) != hipSuccess){
        cerr<<"Compute1D: Error in allocating memory"<<endl;
        cerr<<"Tried to allocate: "<<allocSize<<" bytes"<<endl;
        cerr<<"Size: "<<size<<endl;
        cerr<<"Threads per block: "<<threadsPerBlock<<endl;
        cerr<<"Blocks per grid: "<<blocksPerGrid<<endl;
        cerr<<hipGetErrorString(hipGetLastError())<<endl;
        cerr<<hipGetErrorString(hipPeekAtLastError())<<endl;
        throw invalid_argument("Error in allocating memory");
    }

}

template <typename T>
Compute1D<T>::Compute1D(vector<T> hdata, size_t dataSize) {
    *this = Compute1D(dataSize);
    if (hipMemcpy(this->data, hdata.data(), allocSize, hipMemcpyHostToDevice) != hipSuccess) {
        cout<<"Error in copying data to GPU"<<endl;
        hipFree(this->data);
    }
}

template <typename T>
T* Compute1D<T>::add(BaseCompute<T>& compute) {
    T* c;  
    if(hipMallocManaged(&c, this->size * sizeof(T)) != hipSuccess){
        cout<<"Error in allocating memory"<<endl;
        throw runtime_error("Error in allocating memory");
    }
    addKernel<<<blocksPerGrid, threadsPerBlock>>>(this->data, compute.getData(), c, this->size);

    return c;
}

template <typename T>
T* Compute1D<T>::add(float b) {
    T* c;  
    if(hipMallocManaged(&c, this->size * sizeof(T)) != hipSuccess){
        cout<<"Error in allocating memory"<<endl;
        throw runtime_error("Error in allocating memory");
    }
    addKernel<<<blocksPerGrid, threadsPerBlock>>>(this->data, b, c, size);

    return c;
}

template <typename T>
T* Compute1D<T>::dot(BaseCompute<T>& compute){ 
    T *c; 
    
    if(hipMallocManaged(&c, this->size * sizeof(T)) != hipSuccess){
        cout<<"1dcompute:dot: Error in allocating memory"<<endl;
        throw runtime_error("1dcompute:dot Error in allocating memory");
    }
    mulKernel<<<blocksPerGrid, threadsPerBlock>>>(this->data, compute.getData(), c, size);
    thrust::device_vector<T> d_vec(c, c + size);
    T sum = thrust::reduce(d_vec.begin(), d_vec.end(), 0.0, thrust::plus<float>());
    
    T* out = new T[1];
    if(hipMallocManaged(&out, this->size * sizeof(T)) != hipSuccess){
        cout<<"1dcompute:dot: Error in allocating memory"<<endl;
        throw runtime_error("1dcompute:dot Error in allocating memory");
    }
    out[0] = sum;

    

    hipFree(c);

    // dotKernel<<<blocksPerGrid, threadsPerBlock>>>(this->data, b, c, size);
    // dotKernel2d<<<gridDim, blockDim>>>(this->data, b, c, shape[0], shape[1]);
    
    return out;

}

//TODO: refactor remove unused parameters
template <typename T>
T* Compute1D<T>::dot(float b){ 
    T *c; 
    if(hipMallocManaged(&c, this->size * sizeof(T)) != hipSuccess){
        cout<<"Error in allocating memory"<<endl;
        throw runtime_error("Error in allocating memory");
    }
    dotKernel<<<blocksPerGrid, threadsPerBlock>>>(this->data, b, c, this->size);

    return c;

}

template <typename T>
T* Compute1D<T>::mul(BaseCompute<T>& compute){ 

    T* c; 
    if(hipMallocManaged(&c, this->size * sizeof(T)) != hipSuccess){
        cout<<"Error in allocating memory"<<endl;
        throw runtime_error("Error in allocating memory");
    }
    mulKernel<<<blocksPerGrid, threadsPerBlock>>>(this->data, compute.getData(), c, this->size);

    return c;
}

template <typename T>
T* Compute1D<T>::mul(float b){ 
    T* c; 
    if(hipMallocManaged(&c, this->size * sizeof(T)) != hipSuccess){
        cout<<"Error in allocating memory"<<endl;
        throw runtime_error("Error in allocating memory");
    }
    mulKernel<<<blocksPerGrid, threadsPerBlock>>>(this->data, b, c, this->size);

    return c;
}

template <typename T>
T *Compute1D<T>::greater(BaseCompute<T> &compute)
{
    T* c; 
    if(hipMallocManaged(&c, this->size * sizeof(T)) != hipSuccess){
        cout<<"Error in allocating memory"<<endl;
        throw runtime_error("Error in allocating memory");
    }
    greaterKernel<<<blocksPerGrid, threadsPerBlock>>>(this->data, compute.getData(), c, this->size);
    return c;
}

template <typename T>
T *Compute1D<T>::greater(float b)
{
    T* c; 
    if(hipMallocManaged(&c, this->size * sizeof(T)) != hipSuccess){
        cout<<"Error in allocating memory"<<endl;
        throw runtime_error("Error in allocating memory");
    }
    greaterKernel<<<blocksPerGrid, threadsPerBlock>>>(this->data, b, c, this->size);
    return c;
}

template <typename T>
T *Compute1D<T>::greaterEqual(BaseCompute<T> &compute)
{
    T* c; 
    if(hipMallocManaged(&c, this->size * sizeof(T)) != hipSuccess){
        cout<<"Error in allocating memory"<<endl;
        throw runtime_error("Error in allocating memory");
    }
    greaterEqualKernel<<<blocksPerGrid, threadsPerBlock>>>(this->data, compute.getData(), c, this->size);
    return c;
}

template <typename T>
T *Compute1D<T>::greaterEqual(float b)
{
    T* c; 
    if(hipMallocManaged(&c, this->size * sizeof(T)) != hipSuccess){
        cout<<"Error in allocating memory"<<endl;
        throw runtime_error("Error in allocating memory");
    }
    greaterEqualKernel<<<blocksPerGrid, threadsPerBlock>>>(this->data, b, c, this->size);
    return c;
}

template <typename T>
T *Compute1D<T>::less(BaseCompute<T> &compute)
{
    T* c; 
    if(hipMallocManaged(&c, this->size * sizeof(T)) != hipSuccess){
        cout<<"Error in allocating memory"<<endl;
        throw runtime_error("Error in allocating memory");
    }
    lessKernel<<<blocksPerGrid, threadsPerBlock>>>(this->data, compute.getData(), c, this->size);
    return c;
}

template <typename T>
T *Compute1D<T>::less(float b)
{
    T* c; 
    if(hipMallocManaged(&c, this->size * sizeof(T)) != hipSuccess){
        cout<<"Error in allocating memory"<<endl;
        throw runtime_error("Error in allocating memory");
    }
    lessKernel<<<blocksPerGrid, threadsPerBlock>>>(this->data, b, c, this->size);
    return c;
}

template <typename T>
T *Compute1D<T>::lessEqual(BaseCompute<T> &compute)
{
    T* c; 
    if(hipMallocManaged(&c, this->size * sizeof(T)) != hipSuccess){
        cout<<"Error in allocating memory"<<endl;
        throw runtime_error("Error in allocating memory");
    }
    lessEqualKernel<<<blocksPerGrid, threadsPerBlock>>>(this->data, compute.getData(), c, this->size);
    return c;
}

template <typename T>
T *Compute1D<T>::lessEqual(float b)
{
    T* c; 
    if(hipMallocManaged(&c, this->size * sizeof(T)) != hipSuccess){
        cout<<"Error in allocating memory"<<endl;
        throw runtime_error("Error in allocating memory");
    }
    lessEqualKernel<<<blocksPerGrid, threadsPerBlock>>>(this->data, b, c, this->size);
    return c;
}

template <typename T>
T *Compute1D<T>::equal(BaseCompute<T> &compute)
{
    T* c; 
    if(hipMallocManaged(&c, this->size * sizeof(T)) != hipSuccess){
        cout<<"Error in allocating memory"<<endl;
        throw runtime_error("Error in allocating memory");
    }
    equalKernel<<<blocksPerGrid, threadsPerBlock>>>(this->data, compute.getData(), c, this->size);
    return c;
}

template <typename T>
T *Compute1D<T>::equal(float b)
{
    T* c; 
    if(hipMallocManaged(&c, this->size * sizeof(T)) != hipSuccess){
        cout<<"Error in allocating memory"<<endl;
        throw runtime_error("Error in allocating memory");
    }
    equalKernel<<<blocksPerGrid, threadsPerBlock>>>(this->data, b, c, this->size);
    return c;
}

template <typename T>
T* Compute1D<T>::pow(float n){ 
    T* out; 
    if(hipMallocManaged(&out, this->size * sizeof(T)) != hipSuccess){
        cout<<"Error in allocating memory"<<endl;
        throw runtime_error("Error in allocating memory");
    }
    powKernel<<<blocksPerGrid, threadsPerBlock>>>(this->data, out, n, this->size);
    return out;
}

template <typename T>
T* Compute1D<T>::tanh(){ 
    T* out; 
    if(hipMallocManaged(&out, this->size * sizeof(T)) != hipSuccess){
        cout<<"Error in allocating memory"<<endl;
        throw runtime_error("Error in allocating memory");
    }
    tanhKernel<<<blocksPerGrid, threadsPerBlock>>>(this->data, out, this->size);


    return out;
}

template <typename T>
T *Compute1D<T>::log()
{
    T* out; 
    if(hipMallocManaged(&out, this->size * sizeof(T)) != hipSuccess){
        cout<<"Error in allocating memory"<<endl;
        throw runtime_error("Error in allocating memory");
    }
    logKernel<<<blocksPerGrid, threadsPerBlock>>>(this->data, out, this->size);
    
    return out;
}

template <typename T>
T *Compute1D<T>::exp()
{
    T* out; 
    if(hipMallocManaged(&out, this->size * sizeof(T)) != hipSuccess){
        cout<<"Error in allocating memory"<<endl;
        throw runtime_error("Error in allocating memory");
    }
    expKernel<<<blocksPerGrid, threadsPerBlock>>>(this->data, out, this->size);
    
    return out;

}

template <typename T>
T *Compute1D<T>::sigmoid()
{
    T* out; 
    if(hipMallocManaged(&out, this->size * sizeof(T)) != hipSuccess){
        cout<<"Error in allocating memory"<<endl;
        throw runtime_error("Error in allocating memory");
    }
    sigmoidKernel<<<blocksPerGrid, threadsPerBlock>>>(this->data, out, this->size);
    
    return out;
}

template <typename T>
T *Compute1D<T>::relu()
{
    T* out; 
    if(hipMallocManaged(&out, this->size * sizeof(T)) != hipSuccess){
        cout<<"Error in allocating memory"<<endl;
        throw runtime_error("Error in allocating memory");
    }
    reluKernel<<<blocksPerGrid, threadsPerBlock>>>(this->data, out, this->size);
    return out;
}

template <typename T>
T *Compute1D<T>::sum()
{
    thrust::device_vector<T> d_vec(data, data + this->size);
    T sum = thrust::reduce(d_vec.begin(), d_vec.end(), 0.0, thrust::plus<T>());
    T* out = new T[1];
    if(hipMallocManaged(&out, this->size * sizeof(T)) != hipSuccess){
        cout<<"1dcompute:dot: Error in allocating memory"<<endl;
        throw runtime_error("1dcompute:dot Error in allocating memory");
    }
    out[0] = sum;
    
    return out;
}

template <typename T>
T* Compute1D<T>::sum(int axis)
{
    if(axis != 0){
        throw invalid_argument("Invalid axis for 1D array. Only axis=0 is allowed.");
    }
    return sum();
}
template <typename T>
T *Compute1D<T>::subArray(vector<vector<size_t>> dimRange)
{
    if(dimRange.size() != 1){
        throw invalid_argument("Invalid start index for 1D array. Only 1D start index is allowed.");
    }
    if (dimRange[0].size() != 2)
    {
        throw invalid_argument("Start and end index must be provided for 1D array.");
    }
    T* out;
    size_t newSize = dimRange[0][1] - dimRange[0][0];
    if(hipMallocManaged(&out, newSize * sizeof(T)) != hipSuccess){
        cout<<"Error in allocating memory"<<endl;
        throw runtime_error("Error in allocating memory");
    }
    dim3 threadsPerBlock(256);
    dim3 blocksPerGrid((newSize + threadsPerBlock.x - 1) / threadsPerBlock.x);
    subArrayKernel<<<blocksPerGrid, threadsPerBlock>>>(this->data, out, dimRange[0][0], newSize);
    return out;
}
template <typename T>
void Compute1D<T>::fill(T val)
{
    fillKernel<<<blocksPerGrid, threadsPerBlock>>>(this->data, val, size);

}
template <typename T>
void Compute1D<T>::fillRandom(unsigned int seed){
    fillRandomKernel<<<blocksPerGrid, threadsPerBlock>>>(this->data, size, seed);

}