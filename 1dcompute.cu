#include "hip/hip_runtime.h"
#include <iostream> 
#include "compute.cu"
#include "cuda_compute.cu"
#include <stdexcept>
using namespace std;
template <typename T = double>
class Compute1D : public BaseCompute<T> {
    private: 
        int threadsPerBlock;
        int blocksPerGrid;
        int allocSize;
        long size; 
        T* data;
    public:
        Compute1D() {
        }
        ~Compute1D() {
            cout<<"Freeing memory"<<endl;
            hipFree(this->data);
        }
        T* getData() {
            return this->data;
        }
        void setData(T* data) {
            this->data = data;
        }
        Compute1D(long size){
            data = new T[size];
            this->size = size;
            this->threadsPerBlock = 256;
            this->blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
        
            // Initialize data on GPU
            int allocSize = size * sizeof(T);
            if(hipMallocManaged(&this->data, allocSize) != hipSuccess){
                cout<<"Error in allocating memory"<<endl;
            }
            // debug data 
            // vector<T> cdata(size);
            // if (hipMemcpy(cdata.data(), this->data, allocSize, hipMemcpyDeviceToHost) != hipSuccess) {
            //     cout<<"Error in copying data to GPU"<<endl;
            //     hipFree(this->data);
            // }
            // cout<<"Data copied to GPU"<<endl;
            // cout<<"Data: ";
            // for(int i = 0; i < size; i++){
            //     cout<<cdata[i]<<" ";
            // }
            // cout<<endl;
            hipDeviceSynchronize();
        }
        
        Compute1D(vector<T> hdata, int size) {
            if (hipMemcpy(this->data, hdata.data(), allocSize, hipMemcpyHostToDevice) != hipSuccess) {
                cout<<"Error in copying data to GPU"<<endl;
                hipFree(this->data);
            }
            *this = Compute1D(size);
        }

        T* add(T* b) {
            T* c;  
            if(hipMallocManaged(&c, size * sizeof(T)) != hipSuccess){
                cout<<"Error in allocating memory"<<endl;
                throw runtime_error("Error in allocating memory");
            }
            addKernel<<<blocksPerGrid, threadsPerBlock>>>(this->data, b, c, size);
            hipDeviceSynchronize();
            return c;
        }
        
        T* add(double b) {
            T* c;  
            if(hipMallocManaged(&c, size * sizeof(T)) != hipSuccess){
                cout<<"Error in allocating memory"<<endl;
                throw runtime_error("Error in allocating memory");
            }
            addKernel<<<blocksPerGrid, threadsPerBlock>>>(this->data, b, c, size);
            hipDeviceSynchronize();
            return c;
        }

        T* dot(T * b){ 
            T *c; 
            
            if(hipMallocManaged(&c, size * sizeof(T)) != hipSuccess){
                cout<<"Error in allocating memory"<<endl;
                throw runtime_error("Error in allocating memory");
            }
            dotKernel<<<blocksPerGrid, threadsPerBlock>>>(this->data, b, c, size);
            hipDeviceSynchronize();
            return c;

        }
        T* dot(double b){ 
            T *c; 
            
            if(hipMallocManaged(&c, size * sizeof(T)) != hipSuccess){
                cout<<"Error in allocating memory"<<endl;
                throw runtime_error("Error in allocating memory");
            }
            dotKernel<<<blocksPerGrid, threadsPerBlock>>>(this->data, b, c, size);
            hipDeviceSynchronize();
            return c;

        }
    
        // convert matrix vals to negative 
        T* neg(){ 
            T* out; 
            if(hipMallocManaged(&out, size * sizeof(T)) != hipSuccess){
                cout<<"Error in allocating memory"<<endl;
                throw runtime_error("Error in allocating memory");
            }
            dotKernel<<<blocksPerGrid, threadsPerBlock>>>(this->data, -1.0, out, size);
            hipDeviceSynchronize();
            return out;
        }

        T* pow(double n){ 
            T* out; 
            if(hipMallocManaged(&out, size * sizeof(T)) != hipSuccess){
                cout<<"Error in allocating memory"<<endl;
                throw runtime_error("Error in allocating memory");
            }
            powKernel<<<blocksPerGrid, threadsPerBlock>>>(this->data, out, n, size);
            hipDeviceSynchronize();
            return out;
        }

        T* tanh(){ 
            T* out; 
            if(hipMallocManaged(&out, size * sizeof(T)) != hipSuccess){
                cout<<"Error in allocating memory"<<endl;
                throw runtime_error("Error in allocating memory");
            }
            tanhKernel<<<blocksPerGrid, threadsPerBlock>>>(this->data, out, size);

            hipDeviceSynchronize();
            return out;
        }

        void fill(T val){
            fillKernel<<<blocksPerGrid, threadsPerBlock>>>(this->data, val, size);
            hipDeviceSynchronize();
        }
        void toDevice() {
            hipMemcpy(this->data, this->data, allocSize, hipMemcpyHostToDevice);
            hipDeviceSynchronize();
        }
        void toHost() {
            cout<<"Copy to host"<<endl;
            hipMemcpy(this->data, this->data, allocSize, hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
        }
};